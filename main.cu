#include "hip/hip_runtime.h"
/*******************************************************************
c* Multimodal Deformable Image Registration			   *
c* via Mutual Information or Bhattacharyya Distantce               *
c* Version: 1.0                                                    *
c* Language: C, CUDA                                               *
c*                                                                 *
c* Developer: Yifei Lou                                            *
c* Email: yifei.lou@ece.gatech.edu                                 *
c*                                                                 *
c* School of Electrical and Computer Engineering                   *   
c* Georgia Institute of Technology                                 *
c* Atlanta, GA, 30318                                              *
c* Website: http://groups.bme.gatech.edu/groups/bil/               *
c*                                                                 *
c* Copyright (c) 2011                                              *
c* All rights reserved.                                            *
c*                                                                 *
c* Permission to use, copy, or modify this code and its            *
c* documentation for scientific purpose is hereby granted          *
c* without fee, provided that this copyright notice appear in      *
c* all copies and that both that copyright notice and this         *
c* permission notice appear in supporting documentation. The use   *
c* for commercial purposes is prohibited without permission.       *
c*                                                                 *
c* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND          *
c* CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,     *
c* INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF        *
c* MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE        *
c* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR            *
c* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,    *
c* SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES INCLUDING, BUT NOT *
c* LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF*
c* USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED *
c* AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT     *
c* LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
c* IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF  *
c* THE POSSIBILITY OF SUCH DAMAGE.                                 *
c*                                                                 *
c******************************************************************/

/*******************************************************************
c* Short discription                                               *
c*   main code of the multi-modal deformable registration          *
c*    it calls all the other components                            *
c******************************************************************/



// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <iostream>


// includes, gloable variables
#include "global.h"
#include "convolution.cu"
 
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include <hip/hip_runtime.h>   // for float2

using namespace std;
using namespace thrust;

//	include files
#include "initialize.cu"
#include "funcHistogram.cu"
#include "funcImageDomain.cu"
#include "compute.cu"
#include "finalize.cu"





/****************************************************
	main program
****************************************************/
int main( int argc, char** argv) 
{
	cout << endl << "****************************************" << endl;
        cout << "Computation parameters..." << endl;
        cout << "****************************************" << endl ;

        int device = DEVICENUMBER;
//      device number

        hipSetDevice(device);
        cout << "Using device # " << device << endl;
//      choose which device to use

        hipGetDeviceCount(&deviceCount);
	cout<<"Device count: "<<deviceCount<<endl;
        hipGetDeviceProperties(&dP,device);
        cout<<"Max threads per block: "<<dP.maxThreadsPerBlock<<endl;
        cout<<"Max Threads DIM: "<<dP.maxThreadsDim[0]<<" x "<<dP.maxThreadsDim[1]<<" x "<<dP.maxThreadsDim[2]<<endl;
        cout<<"Max Grid Size: "<<dP.maxGridSize[0]<<" x "<<dP.maxGridSize[1]<<" x "<<dP.maxGridSize[2]<<endl;
	printf("Device %d: \"%s\" with Compute %d.%d capability\n", 
			device, dP.name, dP.major, dP.minor);
//      obtain computing resource


	nblocks_hist.x = NBLOCKX;
        nblocks_hist.y =  ((1 + (nBin*nBin - 1)/NTHREAD_PER_BLOCK) - 1) / NBLOCKX + 1; 

	cout << endl << "****************************************" << endl;
        cout << "Computing starts..." << endl;
        cout << "****************************************" << endl << endl;

//	mark the start total time timer 
	//unsigned int totalTimer = 0;
	StopWatchInterface *totalTimer = NULL;
    	sdkCreateTimer( &totalTimer);
    	sdkStartTimer( &totalTimer);

/******************************************************
	initialize
******************************************************/
	cout << "\n\n";
	cout << "Initializing MI 3Dreg program...\n\n";
	
//////  CBLAS initialization ////////////////////////////

        cout << "Initializing CUBLAS..." << endl;

        hipblasStatus_t status = cublasInit();
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
                fprintf (stderr, "!!!! CUBLAS initialization error\n");
                getchar();
                exit(0);
        }
//      initialize CUBLAS
	
	initData();
	
	initGaussKernel();


	
/******************************************************
	start iterations
******************************************************/
	//unsigned int timer = 0;
	StopWatchInterface *timer = NULL;
        sdkCreateTimer( &timer);
        sdkStartTimer( &timer);
//      mark the start time

        cout << "\n\n";
        cout << "Performing registration...\n\n";

	for(int scale = NSCALE-1; scale >=0; scale--)
	{
		NX = NX0/pow(2, scale);
		NY = NY0/pow(2, scale);
		NZ = (NZ0-1)/pow(2, scale) +1;
	
		sDATA_SIZE = (NX*NY*NZ)* sizeof(float);		

		nblocks.x = NBLOCKX;
        	nblocks.y =  ((1 + (NX*NY*NZ - 1)/NTHREAD_PER_BLOCK) - 1) / NBLOCKX + 1; 
		printf("current scale = %d, size of image = %d x %d x %d ... \n", scale, NX, NY, NZ);
		if(scale<NSCALE-1)
		{
			upSample<<<nblocks, NTHREAD_PER_BLOCK>>>(d_mv_x[scale+1], d_mv_x[scale], NX, NY, NZ);
			upSample<<<nblocks, NTHREAD_PER_BLOCK>>>(d_mv_y[scale+1], d_mv_y[scale], NX, NY, NZ);
			upSample<<<nblocks, NTHREAD_PER_BLOCK>>>(d_mv_z[scale+1], d_mv_z[scale], NX, NY, NZ);
		}
		
		

		compute(d_im_move[scale], d_im_static[scale], d_mv_x[scale], d_mv_y[scale], d_mv_z[scale], MAX_ITER);

		printf("\n\n");
	}


	
	

	

	hipDeviceSynchronize();
	sdkStopTimer( &timer);
        printf("\n\n****************************************\n");
        printf( "Computing time: %f (ms)\n", sdkGetTimerValue( &timer));
        printf("****************************************\n\n\n");
        sdkDeleteTimer( &timer);
//      mark the end timer and print

/******************************************************
	finalize
******************************************************/

	printf("Finalizing program...\n\n");
	
	fina();

/****   shut down CBLAS ********/

        status = cublasShutdown();
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
                fprintf (stderr, "!!!! shutdown error (A)\n");
                getchar();
                exit(0);
        }
//      Shut down CUBLAS

	hipDeviceSynchronize();


//	mark the end total timer
	sdkStopTimer( &totalTimer);
	printf("\n\n****************************************\n");
    	printf( "Entire program time: %f (ms)\n", sdkGetTimerValue( &totalTimer));
    	printf("****************************************\n\n\n");
	sdkDeleteTimer( &totalTimer);


	printf("Have a nice day!\n");
	
    	hipDeviceReset();	
	




    	//cutilExit(argc, argv);
    	exit(0);
	return 0;


}


